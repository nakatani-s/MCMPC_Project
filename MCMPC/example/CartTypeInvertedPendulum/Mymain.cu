/*-- 
--*/
#include <iostream>
#include <stdio.h>
#include <time.h>
#include "../../include/MCMPC_all_Header.cuh"
#include "../../include/Initializer.cuh"
#include "../../include/MCMPC_Controller.cuh"

ControllerInfo _controller;
SpecGPU get_info;
ControllerParams get_param;
DataMessanger *host_, *device_;
InputSequences *Input_Seq;

int main(int argc, char **argv){
    int number;
    time_t now = time(NULL);
    struct tm *pnow = localtime(&now);
    number = pnow->tm_sec;

    float *State, *Input, *Diff_State;
    hiprandState *seedMaker;
    hipMalloc((void**)&seedMaker, NUM_OF_BLOCKS * NUM_OF_THREAD_PER_BLOCKS * DIM_U * sizeof(hiprandState));
    setup_kernel<<<NUM_OF_BLOCKS * DIM_U, NUM_OF_THREAD_PER_BLOCKS>>>(seedMaker, number);
    State = (float*)malloc(DIM_X * sizeof(float));
    Input = (float*)malloc(DIM_U * sizeof(float));
    Diff_State = (float*)malloc(DIM_X * sizeof(float));
    host_ = (DataMessanger*)malloc(NUM_OF_BLOCKS * sizeof(DataMessanger));
    hipMalloc(&device_ , NUM_OF_BLOCKS * sizeof(DataMessanger));
    Input_Seq = (InputSequences*)malloc(DIM_U * sizeof(InputSequences));
    Initialize(State, Input, Diff_State, get_info, get_param, host_, Input_Seq);
    hipMemcpy(device_, host_, NUM_OF_BLOCKS*sizeof(DataMessanger),hipMemcpyHostToDevice);
    get_param.NUM_CYCLES = 0;
    printf("hoge\n");
    float in_h_param[NUM_OF_SYS_PARAMETERS];
    for( int i = 0; i < NUM_OF_SYS_PARAMETERS; i++){
        in_h_param[i] = system_params[i];
    }

    switch(WITH_TERMINAL_COST){
        case 1:
            set_GPU_constant_values(Q, R, system_params, constraint_for_input, constraint_for_state);
            break;
        case 2:
            set_GPU_constant_values_2(Q, R, Qf, Rf, system_params, constraint_for_input, constraint_for_state);
            break;
        default:
            printf("#Fatal error in designing cost function");
            break;
    }
    
    printf("|--------Start Simulation Loop--------|");
    printf("InitValues : %f\n",host_[10].u[1][10]);
    for(int i = 0; i < 100; i++){
        
        MCMPC_Controller(State, _controller, get_info, get_param, host_, device_, Input_Seq, seedMaker);
        hipMemcpy(host_, device_, NUM_OF_BLOCKS*sizeof(DataMessanger),hipMemcpyDeviceToHost);
        printf("InputFromMain : %f CostFromMain: %f Theta: %f x: %f dx: %f dth: %f\n",Input_Seq[0].u[0] ,host_[host_[0].Best_ID].L, State[1], State[0], State[2], State[3]);
        get_param.NUM_CYCLES = i;
        copy_current_input(Input, Input_Seq);
        Runge_kutta_45_for_Secondary_system(State, Input, in_h_param, get_info.RATE_OF_CYCLE);
    }
}
