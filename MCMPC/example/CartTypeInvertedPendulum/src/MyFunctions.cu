#include "hip/hip_runtime.h"
/*-- 
--*/
#include "../../../include/MyFunctions.cuh"

__host__ __device__ float calc_ddx(float *state, float *input, float *param) {
    float a[10];
    /*-- state[DIM_X] = {x, theta, dx, dtheta} --*/ 
    /*-- input[DIM_U] = { u } --*/
    /*-- param[NUM_OF_SYS_PARAMETERS] = {m_c, m_p, l_p, J, myu_c, myu_p, g}--*/ 
    a[0] = param[3] + powf(param[2], 2) * param[1];		//J+l^2*mp
    a[1] = input[0] - state[2] * param[4] + powf(state[3], 2) * param[2] * param[1] * sinf(state[1]);//u-dx*myuc+dtheta^2*l*mp*sin
    a[2] = cosf(state[1]) * param[2] * param[1];						//cos*l*mp
    a[3] = state[3] * param[5] - param[6] * param[2] * param[1] * sinf(state[1]);//dtheta*myup-g*l*mp*sin
    a[4] = -(a[0] * a[1] + a[2] * a[3]);

    a[5] = powf(cosf(state[1]), 2) * powf(param[2], 2) * powf(param[1], 2);//cos^2 * l_p^2 * m_p^2
    a[6] = param[0] + param[1];		//m_c + m_p
    a[7] = param[3] + powf(param[2], 2) * param[1];		//J+l^2*m_p
    a[8] = a[5] - (a[6] * a[7]);

    return a[4] / a[8]; //current ddx
}

__host__ __device__ float calc_ddtheta(float *state, float *input, float *param) {
    float a[10];
    /*-- state[DIM_X] = {x, theta, dx, dtheta} --*/ 
    /*-- input[DIM_U] = { u } --*/
    /*-- param[NUM_OF_SYS_PARAMETERS] = {m_c, m_p, l_p, J, myu_c, myu_p, g}--*/ 
    a[0] = cosf(state[1]) * param[2] * param[1];		//cos*l*mp
    a[1] = input[0] - state[2] * param[4] + powf(state[1], 2) * param[2] * param[1] * sinf(state[1]);//u-dx*myuc+dtheta^2*l*mp*sin
    a[2] = param[0] + param[1];		//mc+mp
    a[3] = state[3] * param[5] - param[6] * param[2] * param[1] * sinf(state[1]);//dtheta*myup-g*l*mp*sin
    a[4] = -(a[0] * a[1] + a[2] * a[3]);

    a[5] = param[3] * (param[0] + param[1]);		//J(mc+mp)
    a[6] = powf(param[2], 2) * param[1];		//l^2*mp
    a[7] = param[0] + param[1] - powf(cosf(state[1]), 2) * param[1];//mc+mp-cos^2*mp
    a[8] = a[5] + a[6] * a[7];
    return a[4] / a[8];  // current ddtheta
}


/*-- The following formulas are the minimum required --*/ 
__host__ __device__ void get_current_diff_state(float *state, float *input, float *param, float *diff_state){
    /*-- p_state[DIM_X] = {dx, dtheta, ddx, ddtheta}--*/
    /*-- state[DIM_X] = {x, theta, dx, dtheta} --*/ 
    /*-- input[DIM_U] = { u } --*/
    /*-- param[NUM_OF_SYS_PARAMETERS] = {m_c, m_p, l_p, J, myu_c, myu_p, g}--*/
    diff_state[0] = state[2];
    diff_state[1] = state[3];
    diff_state[2] = calc_ddx(state, input, param);
    diff_state[3] = calc_ddtheta(state, input, param);
}

__host__ __device__ float get_stage_cost(float *state, float *diff_state, float *input, float *q, float *r, float *state_constraint){

    float re = 0;
    float theta = state[1];
    while (theta > M_PI)
        theta -= (2 * M_PI);
    while (theta < -M_PI)
        theta += (2 * M_PI);
     
    // re = x*Q11*x + 
    re += state[0] * state[0] * q[0]; // x * x * Q11
    re += theta * theta * q[1];       // th * th * Q22
    re += state[2] * state[2] * q[2]; // dx * dx * Q33
    re += state[3] * state[3] * q[3]; // dth * dth * Q44
    re += input[0] * input[0] * r[0]; // u * u * R

    re += state_constraint_barrier(state[0], state_constraint[0], state_constraint[1], 1);
    // state_constraint_barrier1(state);
    return re;

}