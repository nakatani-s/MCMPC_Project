#include "hip/hip_runtime.h"
/*-- 
--*/
#include "../include/generate_random_input.cuh"

/*#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)*/
    
__global__ void setup_kernel(hiprandState *state,int seed) {
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence number,
     no offset */
    hiprand_init(seed, id, 0, &state[id]);
}

__device__ float generate_u(unsigned int id, hiprandState *state, float ave, float vr) {
    float u;
    //	printf("id:%d",id);

    //For Efficiency
    hiprandState localState = state[id];

    u = hiprand_normal(&localState) * vr + ave;

    return u;
}
