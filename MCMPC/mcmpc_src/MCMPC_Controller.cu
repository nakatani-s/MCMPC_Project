#include "hip/hip_runtime.h"
/*--
    This File include
--*/
#include <time.h>
#include "../include/MCMPC_Controller.cuh"



void set_GPU_constant_values(const float *q, const float *r, const float *sys_pr, const float *inp_cntrnt, const float *st_cntrnt){
    hipMemcpyToSymbol(HIP_SYMBOL(d_Q), &q, DIM_Q * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_R), &r, DIM_R * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_param), &sys_pr, NUM_OF_SYS_PARAMETERS * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(output_constraint), &st_cntrnt, NUM_OF_S_CONSTRAINT * sizeof(float));
    printf("#Successful copy of parameters from host to device d_param == %f\n",sys_pr[0]);
}

void set_GPU_constant_values_2(const float *q, const float *r, const float *qf, const float *rf, const float *sys_pr, const float *inp_cntrnt, const float *st_cntrnt){
    hipMemcpyToSymbol(HIP_SYMBOL(d_Q), &q, DIM_Q * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_R), &r, DIM_R * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_param), &sys_pr, NUM_OF_SYS_PARAMETERS * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(input_constraint), &inp_cntrnt, NUM_OF_I_CONSTRAINT * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(output_constraint), &st_cntrnt, NUM_OF_S_CONSTRAINT * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_Qf), &qf, DIM_Qf * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_Rf), &rf, DIM_Rf * sizeof(float));
    printf("#Successful copy of parameters from host to device");
}



__device__ float generate_u1(unsigned int id, hiprandState *state, float ave, float vr) {
    float u;
    hiprandState localState = state[id];

    u = hiprand_normal(&localState) * vr + ave;

    return u;
}


__global__ void MCMPC_GPU(float *h_state, SpecGPU gpu_info, hiprandState *devSt, DataMessanger *dvc, float *var, InputSequences *InpSeq){
    unsigned int id = threadIdx.x + blockDim.x * blockIdx.x;
    float U_dev[DIM_U][NUM_OF_HORIZON], Input_here[DIM_U];
    float Dev_State[DIM_X], dev_Diff_State[DIM_X];
    float Cost = 0;
    general_copy(Dev_State, h_state, DIM_X);
    for(int i = 0; i < NUM_OF_HORIZON; i++){
        for(int k = 0; k < DIM_U; k++){
            U_dev[k][i] = generate_u1(id, devSt, dvc[blockIdx.x].u[k][i], st_dev[k]); //入力を生成する関数はここ（同じファイル）に記述しないと機能しない
            Input_here[k] = generate_u1(id, devSt, 0, st_dev[k]);
        }
        // update predictive model by using random input
    
        get_current_diff_state(Dev_State, Input_here, d_param, dev_Diff_State);
        //printf("%f %f %f\n", Dev_State[0], Input_here[0],d_param[0]);
        euler_integrator_in_thread(Dev_State, dev_Diff_State, gpu_info.RATE_OF_CYCLE);
        Cost += get_stage_cost(Dev_State ,dev_Diff_State, Input_here, d_Q, d_R, output_constraint);
    }
    //printf("ID:%d Cost:%f\n",id, Cost);
    float exp_COST, S;
    S = Cost / gpu_info.LAMBDA;
    exp_COST  = expf(-S);
    thread_COST[threadIdx.x] = Cost;
    thread_exp_COST[threadIdx.x] = exp_COST;
    __syncthreads();
    if(threadIdx.x == 0){
        NO1 = 0;
        for(int i = 1; i < blockDim.x; i++){
            if(thread_COST[i] < thread_COST[NO1])
                NO1 = i;
        }
    }
    __syncthreads();
    if(threadIdx.x == NO1){
        dvc[blockIdx.x].L = thread_COST[NO1];
        dvc[blockIdx.x].W = thread_exp_COST[NO1];
        for(int i = 0; i < NUM_OF_HORIZON; i++){
            for(int k = 0; k < DIM_U; k++){
                dvc[blockIdx.x].u[k][i] = U_dev[k][i];
            }
        }
	//printf("ID: %d Value: %f\n", id, U_dev[0][10]);
        //dvc[blockIdx.x] = in_block;
    }
} 

void MCMPC_Controller(float *state, float *input, ControllerInfo &info_cont , SpecGPU gpu_info, ControllerParams param, DataMessanger *hst, DataMessanger *dvc, InputSequences *InpSeq, hiprandState *se){
    if(param.NUM_CYCLES == 0){
        hipMemcpyToSymbol(HIP_SYMBOL(d_Q), &Q, DIM_Q * sizeof(float));
        hipMemcpyToSymbol(HIP_SYMBOL(d_R), &R, DIM_R * sizeof(float));
        hipMemcpyToSymbol(HIP_SYMBOL(d_param), &system_params, NUM_OF_SYS_PARAMETERS * sizeof(float));
        hipMemcpyToSymbol(HIP_SYMBOL(output_constraint), &constraint_for_state, NUM_OF_S_CONSTRAINT * sizeof(float));
    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    InputSequences *device_InpSeq;
    float *h_state;
    hipMalloc(&device_InpSeq, DIM_U * sizeof(InputSequences));
    hipMalloc(&h_state,DIM_X * sizeof(float));
    hipMemcpy(h_state, state, DIM_X*sizeof(float), hipMemcpyHostToDevice); //状態量をデバイスで使用する変数にコピー
    
    float variance[DIM_U];
    //variance = (float*)malloc(DIM_U * sizeof(float));
    printf("Function: %f\n", hst[10].u[0][10]);
    /* Iterate Predction Process */
    for(int i = 0; i < gpu_info.ITERATIONS; i++){
        switch(COOLING_PATTERN){
            case 1:
                for(int k = 0; k < DIM_U; k++){
                    variance[k] = geometric_cooling(gpu_info.INIT_VARIANCE[k], i, gpu_info.COOLING_RATES[k]);
                }
                break;
            case 2:
                for(int k = 0; k < DIM_U; k++){
                    variance[k] = hyperbolic_cooling(gpu_info.INIT_VARIANCE[k], i);
                }
                break; 
            default:
                for(int k = 0; k < DIM_U; k++){
                    variance[k] = gpu_info.INIT_VARIANCE[k];
                }
                break;
                
        }
        hipMemcpy(device_InpSeq, InpSeq, DIM_U * sizeof(InputSequences), hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(st_dev), &variance, DIM_U*sizeof(float));
        MCMPC_GPU<<<gpu_info.NUM_BLOCKS,gpu_info.TH_PER_BLS>>>(h_state, gpu_info, se, dvc, variance, InpSeq);
        hipMemcpy(hst, dvc, gpu_info.NUM_BLOCKS * sizeof(DataMessanger),hipMemcpyDeviceToHost); //ここでコピーしても記述されない

        switch(PREDICTIVE_METHOD){
            case 1:
                TOP1_sample_method(hst, gpu_info, InpSeq);
        }
        printf("Values From Function: %f CostFrom: %f  TOP_Input: %f\n", hst[10].u[0][0], hst[10].L, InpSeq[0].u[0]);
    }
    //hst[10].u[0][10] = 1.0;
    //printf("Values From Function: %f %f\n", hst[10].u[0][10], variance[0]);
}
