#include "hip/hip_runtime.h"
/*-- 
--*/
#include "../include/constraint.cuh" 

__host__ __device__ float state_constraint_barrier(float st, float min, float max, int TYPE){
    float add = 0;
    float med = 0;
    switch(CONSTRAINT_TYPE){
        case 1:
            med = (max + min)/2;
            if(med < st){
                add += 1/(powf((max - st),2));
                if(max < st){
                    add += FLT_MAX;
                }
            }else{
                add += 1/(powf((min - st),2));
                if(st < min){
                    add += FLT_MAX;
                }
            }
            break;
        case 2:
            med = (max + min) /2;
            if(med < st){
                add += -logf(max-st);
                if(max < st){
                    add += FLT_MAX;
                }
            }else{
                add += -logf(st - min);
                if(st < min){
                    add += FLT_MAX;
                }
            }
            break;
        default:
            if( st < min ){
                add += FLT_MAX;
            }
            if( max < st){
                add += FLT_MAX;
            }
            break;
    }

    return add;
}

