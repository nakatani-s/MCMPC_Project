#include "hip/hip_runtime.h"
/*-- 
--*/
#include "../include/cooling_method.cuh"

float geometric_cooling(float init_st, int NUM_ITER, float rate){
    float ret;
    ret = powf(rate, NUM_ITER) * init_st;
    return ret;
}

float hyperbolic_cooling(float init_st, int NUM_ITER){
    float ret;
    ret = init_st / sqrtf(NUM_ITER + 1);
    return ret;
}